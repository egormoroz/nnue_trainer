
#include <hip/hip_runtime.h>
using uint32_t = unsigned int;
using int32_t = int;

extern "C" __global__
void feature_transformer_slice_backward(
    const int32_t* const feature_indices,
    const float*   const feature_values,
          float*   const weight_grad,
          float*   const bias_grad,
    const float*   const output_grad)
{
    const uint32_t max_active_features = 30;
    const uint32_t output_thread_slice_size = 1;
    const uint32_t output_size = 256;

    __shared__
        float shared_output_grad[output_size];

    const uint32_t block_idx = blockIdx.x;
    const uint32_t slice_offset = threadIdx.x * output_thread_slice_size;

    const float* output_grad_slice   = output_grad + block_idx * output_size + slice_offset;
          float* bias_grad_slice     = bias_grad + slice_offset;
          float* shared_output_grad_slice = shared_output_grad + slice_offset;

    const int32_t* feature_index_row = feature_indices + block_idx * max_active_features;
    const float* feature_value_row = feature_values + block_idx * max_active_features;

    #pragma unroll
    for (uint32_t s = 0; s < output_thread_slice_size; ++s)
        shared_output_grad_slice[s] = output_grad_slice[s];

    #pragma unroll
    for (uint32_t s = 0; s < output_thread_slice_size; ++s) {
        const float sog = shared_output_grad_slice[s];
        if (sog != 0.f)
            atomicAdd(&bias_grad_slice[s], sog);
    }

    for (uint32_t k = 0; k < max_active_features; ++k) {
        const int32_t feature_index = feature_index_row[k];
        const float   feature_value = feature_value_row[k];

        if (feature_index != -1) {
            float* const weight_grad_slice = weight_grad 
                + feature_index * output_size + slice_offset;
            #pragma unroll
            for (uint32_t s = 0; s < output_thread_slice_size; ++s) {
                const float sog = shared_output_grad_slice[s];
                if (sog != 0.f)
                    atomicAdd(&weight_grad_slice[s], sog * feature_value);
            }
        } else break;
    }

}

